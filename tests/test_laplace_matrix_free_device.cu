/*************************************************************************
 * Copyright (c) 2017-2019 by the mfmg authors                           *
 * All rights reserved.                                                  *
 *                                                                       *
 * This file is part of the mfmg libary. mfmg is distributed under a BSD *
 * 3-clause license. For the licensing terms see the LICENSE file in the *
 * top-level directory                                                   *
 *                                                                       *
 * SPDX-License-Identifier: BSD-3-Clause                                 *
 *************************************************************************/

#define BOOST_TEST_MODULE laplace_matrix_free_device

#include <deal.II/base/hip/hip_runtime.h>
#include <deal.II/lac/precondition.h>
#include <deal.II/lac/trilinos_precondition.h>

#include "laplace.hpp"
#include "laplace_matrix_free_device.cuh"
#include "main.cc"

namespace tt = boost::test_tools;

template <int dim>
class ExactSolution : public dealii::Function<dim>
{
public:
  ExactSolution() = default;

  double value(dealii::Point<dim> const &p,
               unsigned int const component = 0) const override;
};

template <int dim>
double ExactSolution<dim>::value(dealii::Point<dim> const &p,
                                 unsigned int const) const
{
  double val = 1.;
  for (unsigned int d = 0; d < dim; ++d)
    val *= (p[d] - 1.) * p[d];

  return val;
}

template <int dim>
class Source
{
public:
  Source() = default;

  template <typename ScalarType>
  ScalarType value(dealii::Point<dim, ScalarType> const &p) const;
};

template <int dim>
template <typename ScalarType>
ScalarType Source<dim>::value(dealii::Point<dim, ScalarType> const &p) const
{
  ScalarType val = 0.;
  for (unsigned int d = 0; d < dim; ++d)
  {
    ScalarType tmp = 1.;
    for (unsigned int i = 0; i < dim; ++i)
      if (i != d)
        tmp *= (p[i] - 1.) * p[i];

    val -= 2. * tmp;
  }

  return val;
}

template <int dim>
class MaterialProperty
{
public:
  MaterialProperty() = default;

  template <typename ScalarType>
  dealii::VectorizedArray<ScalarType>
  value(dealii::Point<dim, dealii::VectorizedArray<ScalarType>> const &p) const;
};

template <int dim>
template <typename ScalarType>
dealii::VectorizedArray<ScalarType> MaterialProperty<dim>::value(
    dealii::Point<dim, dealii::VectorizedArray<ScalarType>> const &) const
{
  return dealii::make_vectorized_array<ScalarType>(1.);
}

template <int dim>
class MSource : public dealii::Function<dim>
{
public:
  MSource() = default;

  double value(dealii::Point<dim> const &p,
               unsigned int const component = 0) const override;
};

template <int dim>
double MSource<dim>::value(dealii::Point<dim> const &p,
                           unsigned int const) const
{
  double val = 0.;
  for (unsigned int d = 0; d < dim; ++d)
  {
    double tmp = 1.;
    for (unsigned int i = 0; i < dim; ++i)
      if (i != d)
        tmp *= (p[i] - 1.) * p[i];

    val += -2. * tmp;
  }

  return val;
}

template <typename VectorType>
class MyCG : public dealii::SolverCG<VectorType>
{
public:
  MyCG(dealii::SolverControl &cn) : dealii::SolverCG<VectorType>(cn) {}

  void print_vectors(const unsigned int step, const VectorType &x,
                     const VectorType &r, const VectorType &d) const override
  {
    //  std::cout << "step " << step << std::endl;
    //  x.print(std::cout);
    //  r.print(std::cout);
    //  d.print(std::cout);
  }
};

BOOST_AUTO_TEST_CASE(laplace_2d)
{
  int constexpr dim = 2;
  int constexpr fe_degree = 3;

  dealii::Utilities::CUDA::Handle cuda_handle;
  Source<dim> source;

  LaplaceMatrixFreeDevice<dim, fe_degree, double> laplace_dev(MPI_COMM_WORLD);
  MaterialProperty<dim> material_property;
  laplace_dev.setup_system(boost::property_tree::ptree(), material_property);
  laplace_dev.assemble_rhs(source);

  dealii::PreconditionIdentity preconditioner;
  laplace_dev.solve(preconditioner);

  // The exact solution is quadratic so the error should be zero.
  ExactSolution<dim> exact_solution;
  BOOST_TEST(laplace_dev.compute_error(exact_solution) == 0.,
             tt::tolerance(1e-14));
}

BOOST_AUTO_TEST_CASE(laplace_3d)
{
  int constexpr dim = 3;
  int constexpr fe_degree = 2;

  dealii::Utilities::CUDA::Handle cuda_handle;
  Source<dim> source;

  LaplaceMatrixFreeDevice<dim, fe_degree, double> laplace_dev(MPI_COMM_WORLD);
  MaterialProperty<dim> material_property;
  laplace_dev.setup_system(boost::property_tree::ptree(), material_property);
  laplace_dev.assemble_rhs(source);
  std::cout << "rhs " << laplace_dev._system_rhs.l2_norm() << std::endl;
  auto solution = laplace_dev._system_rhs;

  dealii::PreconditionIdentity preconditioner;
  laplace_dev.solve(preconditioner);

  // The exact solution is quadratic so the error should be zero.
  ExactSolution<dim> exact_solution;
  BOOST_TEST(laplace_dev.compute_error(exact_solution) == 0.,
             tt::tolerance(1e-14));
}
