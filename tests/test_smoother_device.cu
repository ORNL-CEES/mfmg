#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2017-2019 by the mfmg authors                           *
 * All rights reserved.                                                  *
 *                                                                       *
 * This file is part of the mfmg libary. mfmg is distributed under a BSD *
 * 3-clause license. For the licensing terms see the LICENSE file in the *
 * top-level directory                                                   *
 *                                                                       *
 * SPDX-License-Identifier: BSD-3-Clause                                 *
 *************************************************************************/

#define BOOST_TEST_MODULE smoother_device

#include <mfmg/common/exceptions.hpp>
#include <mfmg/cuda/cuda_matrix_operator.cuh>
#include <mfmg/cuda/cuda_smoother.cuh>
#include <mfmg/cuda/sparse_matrix_device.cuh>
#include <mfmg/cuda/utils.cuh>

#include <deal.II/lac/precondition.h>
#include <deal.II/lac/sparse_matrix.h>
#include <deal.II/lac/sparsity_pattern.h>

#include <boost/property_tree/ptree.hpp>

#include "main.cc"

BOOST_AUTO_TEST_CASE(smoother)
{
  // Create the cusparse handle
  hipsparseHandle_t cusparse_handle = nullptr;
  hipsparseStatus_t cusparse_error_code;
  cusparse_error_code = hipsparseCreate(&cusparse_handle);
  mfmg::ASSERT_CUSPARSE(cusparse_error_code);

  // Create the matrix on the host.
  dealii::SparsityPattern sparsity_pattern;
  dealii::SparseMatrix<double> matrix;
  unsigned int const size = 30;
  std::vector<std::vector<unsigned int>> column_indices(size);
  for (unsigned int i = 0; i < size; ++i)
  {
    unsigned int j_max = std::min(size, i + 2);
    unsigned int j_min = (i == 0) ? 0 : i - 1;
    for (unsigned int j = j_min; j < j_max; ++j)
      column_indices[i].emplace_back(j);
  }
  sparsity_pattern.copy_from(size, size, column_indices.begin(),
                             column_indices.end());
  matrix.reinit(sparsity_pattern);
  for (unsigned int i = 0; i < size; ++i)
  {
    unsigned int j_max = std::min(size - 1, i + 1);
    unsigned int j_min = (i == 0) ? 0 : i - 1;
    matrix.set(i, j_min, -1.);
    matrix.set(i, j_max, -1.);
    matrix.set(i, i, 4.);
  }

  double constexpr scalar_value = 1.;
  std::vector<double> domain_host(size);
  for (auto &v : domain_host)
    v = scalar_value;
  std::vector<double> range_host(size, 0.);
  dealii::Vector<double> domain_vector(size);
  dealii::Vector<double> range_vector(size);
  for (auto &v : domain_vector)
    v = scalar_value;

  // Compute the reference solution
  dealii::PreconditionJacobi<dealii::SparseMatrix<double>> precondition;
  precondition.initialize(matrix);
  dealii::Vector<double> res(domain_vector);
  matrix.vmult(res, range_vector);
  res.add(-1., domain_vector);
  dealii::Vector<double> tmp(range_vector);
  precondition.vmult(tmp, res);
  range_vector.add(-1., tmp);

  // Move the matrix to the device
  auto matrix_dev = std::make_shared<mfmg::SparseMatrixDevice<double>>(
      mfmg::convert_matrix(matrix));
  matrix_dev->cusparse_handle = cusparse_handle;
  cusparse_error_code = hipsparseCreateMatDescr(&matrix_dev->descr);
  mfmg::ASSERT_CUSPARSE(cusparse_error_code);
  cusparse_error_code =
      hipsparseSetMatType(matrix_dev->descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  mfmg::ASSERT_CUSPARSE(cusparse_error_code);
  cusparse_error_code =
      hipsparseSetMatIndexBase(matrix_dev->descr, HIPSPARSE_INDEX_BASE_ZERO);
  mfmg::ASSERT_CUSPARSE(cusparse_error_code);

  // Build the smoother operator
  std::shared_ptr<mfmg::Operator<dealii::LinearAlgebra::distributed::Vector<
      double, dealii::MemorySpace::CUDA>>>
  cuda_op(
      new mfmg::CudaMatrixOperator<dealii::LinearAlgebra::distributed::Vector<
          double, dealii::MemorySpace::CUDA>>(matrix_dev));
  auto param = std::make_shared<boost::property_tree::ptree>();
  mfmg::CudaSmoother<dealii::LinearAlgebra::distributed::Vector<
      double, dealii::MemorySpace::CUDA>>
      smoother_operator(cuda_op, param);

  // Apply the smoother
  auto domain_dev = cuda_op->build_domain_vector();
  auto range_dev = cuda_op->build_range_vector();
  mfmg::cuda_mem_copy_to_dev(domain_host, domain_dev->get_values());
  mfmg::cuda_mem_copy_to_dev(range_host, range_dev->get_values());
  smoother_operator.apply(*domain_dev, *range_dev);

  // Compare the solution
  mfmg::cuda_mem_copy_to_host(range_dev->get_values(), range_host);
  for (unsigned int i = 0; i < size; ++i)
    BOOST_CHECK_CLOSE(range_host[i], range_vector[i], 1e-12);

  // Destroy the cusparse handle
  cusparse_error_code = hipsparseDestroy(cusparse_handle);
  mfmg::ASSERT_CUSPARSE(cusparse_error_code);
}
